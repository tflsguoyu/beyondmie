#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017, Amos Egel (KIT), Lorenzo Pattelli (LENS)
 *                     Giacomo Mazzamuto (LENS)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *  * Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 *
 *  * Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *
 *  * Neither the name of the copyright holder nor the names of its
 *    contributors may be used to endorse or promote products derived from
 *    this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"
#include <math.h>
#include "hip/hip_runtime_api.h"

__device__ float assocLegendreFunction(int const l, int const m, float const ct, float const st, float const *plm_coeffs)
{
	float Plm = 0.0f;
	int jj=0;
	for (int lambda=l-m; lambda>=0; lambda-=2)
	{
		Plm = Plm + pow(st,m) * pow(ct,lambda) * plm_coeffs[jj*(2*LMAX+1)*(2*LMAX+1)+m*(2*LMAX+1)+l];
		jj++;
	}
	return Plm;
}

__device__ float sphericalBesselLookup(int const p, float const r, float const *spjTable, float const rResol)
{
	float spj = 0.0f;
	float rPos = r/rResol;
	int rIdx = int(rPos);    						// points to table position -1, because for each p, the first entry with respect to r in the spjTable is copied 
	rPos -= rIdx; 							 	// (remainder of r/rResol) / rResol
	float rPos2 = pow(rPos,2);
	float rPos3 = pow(rPos,3);
	spj = ((-rPos3+2*rPos2-rPos) * spjTable[rIdx*(2*LMAX+1)+p]
			+ (3*rPos3-5*rPos2+2) * spjTable[(rIdx+1)*(2*LMAX+1)+p]
			+ (-3*rPos3+4*rPos2+rPos) * spjTable[(rIdx+2)*(2*LMAX+1)+p]
			+ (rPos3-rPos2) * spjTable[(rIdx+3)*(2*LMAX+1)+p])/2;
	return spj;
}


__global__ void translationMatrixProduct(int const s2, int const NS, float const *sPosArray,
										float const *sphericalBesselTable, float const *sphericalNeumannTable, float rResol,
										float const *plm_coeffs, float const *re_abTable, float const *im_abTable,
										float const *re_x, float const *im_x, float *re_Wx, float *im_Wx)
{
  int const s1 = blockDim.x * blockIdx.x + threadIdx.x + 1; // receiving sphere number (1...NS)
  float x21, y21, z21;
	float r, cosTheta, sinTheta, phi;
	float re_h[2*LMAX+1];
	float im_h[2*LMAX+1];
	float Ppdm[(2*LMAX+1)*(2*LMAX+2)/2];
	float cosmphi[4*LMAX+1];
	float sinmphi[4*LMAX+1];
	int n1, n2, deltam;
	float re_xTmp, im_xTmp;
	int loopCounter = 0;
	int WxIdx, xIdx, abIdx;
	float re_incr, im_incr;
	float re_abP, im_abP, re_abPh, im_abPh, re_abPheimp, im_abPheimp;
	
  if ((s1!=s2)&&s1<=NS)
	{
		// relative position
		
		x21 = sPosArray[3*(s1-1)]-sPosArray[3*(s2-1)];
		y21 = sPosArray[3*(s1-1)+1]-sPosArray[3*(s2-1)+1];
		z21 = sPosArray[3*(s1-1)+2]-sPosArray[3*(s2-1)+2];
		
		r = sqrt(x21*x21+y21*y21+z21*z21);
		cosTheta = z21/r;
		sinTheta = sqrt(1-cosTheta*cosTheta);
		phi = atan2(y21,x21);
		
		for (int p=0; p<=2*LMAX; p++)	// precompute spherical Hankel functions and Legendre functions
		{
			re_h[p] = sphericalBesselLookup(p,r,sphericalBesselTable,rResol);
			im_h[p] = sphericalBesselLookup(p,r,sphericalNeumannTable,rResol);
			for (int absdm=0; absdm<=p; absdm++)
			{
				Ppdm[p*(p+1)/2+absdm] = assocLegendreFunction(p,absdm,cosTheta,sinTheta,plm_coeffs);
			}
		}
		
		for (int dm=-2*LMAX; dm<=2*LMAX; dm++) // precompute exp(i(m-m')phi)
		{
			cosmphi[dm+2*LMAX] = cosf(dm*phi);
			sinmphi[dm+2*LMAX] = sinf(dm*phi);
		}
		
		for (int tau1=1; tau1<=2; tau1++) // evaluate matrix vector product
		{
			int temp1 = (tau1-1)*LMAX*(LMAX+2);
			for (int l1=1; l1<=LMAX; l1++)
			{
				int coeff1 = temp1+(l1-1)*(l1+1)+l1+1;
				for (int m1=-l1; m1<=l1; m1++)
				{
					n1 = coeff1+m1;
					WxIdx = (n1-1)*NS+s1-1;
					re_incr = 0.0f;
					im_incr = 0.0f;
					
					for (int tau2=1; tau2<=2; tau2++)
					{
						int temp2 = (tau2-1)*LMAX*(LMAX+2);
						for (int l2=1; l2<=LMAX; l2++)
						{
							int coeff2 = temp2+(l2-1)*(l2+1)+l2+1;
							for (int m2=-l2; m2<=l2; m2++)
							{
								n2 = coeff2+m2;
								xIdx = (n2-1)*NS+s2-1;
								re_xTmp = re_x[xIdx];
								im_xTmp = im_x[xIdx];
								deltam=m2-m1;
								abIdx=deltam+2*LMAX;
								for (int p=max(abs(deltam),abs(l1-l2)+abs(tau1-tau2)); p<=l1+l2; p++)
								{
									re_abP = re_abTable[loopCounter]*Ppdm[p*(p+1)/2+abs(deltam)];
									im_abP = im_abTable[loopCounter]*Ppdm[p*(p+1)/2+abs(deltam)];
									
									re_abPh = re_abP*re_h[p] - im_abP*im_h[p];
									im_abPh = re_abP*im_h[p] + im_abP*re_h[p];
									
									re_abPheimp = re_abPh*cosmphi[abIdx] - im_abPh*sinmphi[abIdx];
									im_abPheimp = re_abPh*sinmphi[abIdx] + im_abPh*cosmphi[abIdx];
									
									re_incr += re_abPheimp*re_xTmp - im_abPheimp*im_xTmp;
									im_incr += re_abPheimp*im_xTmp + im_abPheimp*re_xTmp;
									
									loopCounter++;
								}	
							}
						}
					}//tau2
					re_Wx[WxIdx] += re_incr;
					im_Wx[WxIdx] += im_incr;
				} 
			}  
		}//tau1
	}  
}


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	/* input arguments: 
	real_x
	imag_x 
	real_h			
	imag_h	
	Plm_coeffs
	real_ab5
	imag_ab5
	spos
	NS 			
	rResol */

	// initialize the MathWorks GPU API.
	mxInitGPU();
		
	// check number of arguments:
	if (nrhs!=10) {mexErrMsgTxt("wrong number of input arguments");}
	if (nlhs!=2) {mexErrMsgTxt("wrong number of output arguments");}

	// check for GPUArrays
	if (!(mxIsGPUArray(prhs[0]))) {mexErrMsgTxt("real_x is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[1]))) {mexErrMsgTxt("imag_x is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[2]))) {mexErrMsgTxt("real_h is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[3]))) {mexErrMsgTxt("imag_h is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[4]))) {mexErrMsgTxt("Plm_coeffs is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[5]))) {mexErrMsgTxt("real_ab5 is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[6]))) {mexErrMsgTxt("imag_ab5 is not a gpuArray");}
	if (!(mxIsGPUArray(prhs[7]))) {mexErrMsgTxt("spos is not a gpuArray");}
	
	// initialize mxGPUArrays
	mxGPUArray const *mx_real_x = mxGPUCreateFromMxArray(prhs[0]);	
	mxGPUArray const *mx_imag_x = mxGPUCreateFromMxArray(prhs[1]);	
	mxGPUArray const *mx_real_h = mxGPUCreateFromMxArray(prhs[2]);	
	mxGPUArray const *mx_imag_h = mxGPUCreateFromMxArray(prhs[3]);	
	mxGPUArray const *mx_PlmCoeff = mxGPUCreateFromMxArray(prhs[4]);	
	mxGPUArray const *mx_real_ab5 = mxGPUCreateFromMxArray(prhs[5]);
	mxGPUArray const *mx_imag_ab5 = mxGPUCreateFromMxArray(prhs[6]);
	mxGPUArray const *mx_sPos = mxGPUCreateFromMxArray(prhs[7]);
	// mxGPUArray *mx_real_Wx = mxGPUCopyGPUArray(mx_real_x);  // writable copy of real_x ... to be overwritten in kernel
	// mxGPUArray *mx_imag_Wx = mxGPUCopyGPUArray(mx_imag_x);

	mxGPUArray *mx_real_Wx = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(mx_real_x),
                            mxGPUGetDimensions(mx_real_x),
                            mxSINGLE_CLASS,mxREAL,MX_GPU_INITIALIZE_VALUES);
																
	mxGPUArray *mx_imag_Wx = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(mx_real_x),
                            mxGPUGetDimensions(mx_real_x),
                            mxSINGLE_CLASS,mxREAL,MX_GPU_INITIALIZE_VALUES);
	
	// check data types
	if (mxGPUGetClassID(mx_real_x) != mxSINGLE_CLASS) {mexErrMsgTxt("real_x is not single");}
	if (mxGPUGetClassID(mx_imag_x) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_x is not single");}
	if (mxGPUGetClassID(mx_real_h) != mxSINGLE_CLASS) {mexErrMsgTxt("real_h is not single");}
	if (mxGPUGetClassID(mx_imag_h) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_h is not single");}
	if (mxGPUGetClassID(mx_PlmCoeff) != mxSINGLE_CLASS) {mexErrMsgTxt("PlmCoeff is not single");}
	if (mxGPUGetClassID(mx_real_ab5) != mxSINGLE_CLASS) {mexErrMsgTxt("real_ab5 is not single");}
	if (mxGPUGetClassID(mx_imag_ab5) != mxSINGLE_CLASS) {mexErrMsgTxt("imag_ab5 is not single");}
	if (mxGPUGetClassID(mx_sPos) != mxSINGLE_CLASS) {mexErrMsgTxt("sPos is not single");}
	if (mxGetClassID(prhs[8]) != mxINT32_CLASS) {mexErrMsgTxt("NS is not int32");}
	if (mxGetClassID(prhs[9]) != mxSINGLE_CLASS) {mexErrMsgTxt("rResol is not single");}
	
	// initialize host variables
	int const *NS = (int*)mxGetData(prhs[8]);	// total number of spheres
	float const *rResol = (float*)mxGetData(prhs[9]);	// maximal polar quantum number
	int const 	threadsPerBlock = 256;
	int const blocksPerGrid = (NS[0] + threadsPerBlock - 1) / threadsPerBlock;

	// initialize device variables
	float	 	*d_real_Wx = (float *)(mxGPUGetData(mx_real_Wx));
	float	 	*d_imag_Wx = (float *)(mxGPUGetData(mx_imag_Wx));
	float const *d_real_x = (float const *)(mxGPUGetDataReadOnly(mx_real_x));
	float const *d_imag_x = (float const *)(mxGPUGetDataReadOnly(mx_imag_x));
	float const *d_real_h = (float const *)(mxGPUGetDataReadOnly(mx_real_h));
	float const *d_imag_h = (float const *)(mxGPUGetDataReadOnly(mx_imag_h));
	float const *d_PlmCoeff = (float const *)(mxGPUGetDataReadOnly(mx_PlmCoeff));
	float const *d_real_ab5 = (float const *)(mxGPUGetDataReadOnly(mx_real_ab5));
	float const *d_imag_ab5 = (float const *)(mxGPUGetDataReadOnly(mx_imag_ab5));
	float const *d_sPos = (float const *)(mxGPUGetDataReadOnly(mx_sPos));

	
	// float check_re;
	// float check_im;
	
	// start computation
	for (int s2=1; s2<=NS[0]; s2++)
	{
		translationMatrixProduct<<< blocksPerGrid,threadsPerBlock >>> (s2,NS[0],d_sPos,
																		d_real_h, d_imag_h, rResol[0],
																		d_PlmCoeff, d_real_ab5, d_imag_ab5,
																		d_real_x, d_imag_x, d_real_Wx, d_imag_Wx);
		// hipMemcpy(&check_re,d_real_Wx,sizeof(check_re),hipMemcpyDeviceToHost);
		// hipMemcpy(&check_im,d_imag_Wx,sizeof(check_im),hipMemcpyDeviceToHost);
		// mexPrintf("%f %f\n",check_re,check_im);
																																
	}

	// wrap the result up as a MATLAB gpuArray for return
	plhs[0] = mxGPUCreateMxArrayOnGPU(mx_real_Wx);
	plhs[1] = mxGPUCreateMxArrayOnGPU(mx_imag_Wx);

	// destroy mxgpuarrays
	mxGPUDestroyGPUArray(mx_real_x);
	mxGPUDestroyGPUArray(mx_imag_x);
	mxGPUDestroyGPUArray(mx_real_h);
	mxGPUDestroyGPUArray(mx_imag_h);
	mxGPUDestroyGPUArray(mx_PlmCoeff);
	mxGPUDestroyGPUArray(mx_real_ab5);
	mxGPUDestroyGPUArray(mx_imag_ab5);
	mxGPUDestroyGPUArray(mx_sPos);
	mxGPUDestroyGPUArray(mx_real_Wx);
	mxGPUDestroyGPUArray(mx_imag_Wx);
		
	//hipDeviceReset();  // necessary for profiling
}
